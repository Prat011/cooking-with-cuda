#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__constant__ char d_message[20];

__global__ void add_subtract(int* a, int* b, int* c){
    *c = *a+*b+(*a-*b);
    printf("%d",*c);
}

int main() {
    int a, b, c;
    int *d_a, *d_b, *d_c;
    a = 9;
    b = 3;
    int size = sizeof(int);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    calculator<<<1,1>>>(d_a, d_b, d_c);
    hipMemcpy(d_c, &c, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("Hello world from the CPU\n");
}
