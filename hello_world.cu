#include <iostream>
#include <hip/hip_runtime.h>

__constant__ char d_message[20];

__global__ void hello_world(){
    printf("Hello world, from the GPU\n");
}

int main() {
    int threadsperBlock = 256;
    hello_world<<<1,1>>>();
    hipDeviceSynchronize(); //This ensures the code works synchronously
    //without the synchronize function, the CPU print statement executes before the GPU print statement
    printf("Hello world from the CPU\n");
}
